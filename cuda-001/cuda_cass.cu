/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

Base::Base(){

}

Base::~Base(){

}

Derived1::Derived1(){

}

Derived1::~Derived1(){

}

void Derived1::print(){
	cout << "DERIVED1" << std::endl;

}

Derived2::Derived2(){
	cout << "DERIVED2" << std::endl;

}

Derived2::~Derived2(){

}

void Derived2::print(){

}

Container::Container(){
	bases = NULL;
	d1 = NULL;
	d2 = NULL;

}

Container::~Container(){

}
