/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

Base::Base(){

}

Base::~Base(){

}

Derived1::Derived1(){

}

Derived1::~Derived1(){

}

Container::Container(){
	bases = NULL;
	derivations = NULL;

}

Container::~Container(){

}
