#include "hip/hip_runtime.h"
// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

// Defs
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

__global__ void KERNEL_SETUP(Container *container){

}

__global__ void KERNEL_MAIN(Container *container){

}

int main(int argc, const char * argv[]){
	hipError_t result = hipSuccess;
	const char *resultString;

	Container *c;

	hipMalloc((void**)&c, sizeof(Container));
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	KERNEL_SETUP<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	KERNEL_MAIN<<<16,16>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	if(c != NULL) hipFree(c);
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	// get outta here
	hipDeviceReset();
	return EXIT_SUCCESS;

}
