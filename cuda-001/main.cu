#include "hip/hip_runtime.h"
// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

// Defs
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif


__global__ void KERNEL_SETUP(Container *container){
	container->init(5,5);

}

__global__ void KERNEL_CLEANUP(Container *container){
	container->cleanup();
}

__global__ void KERNEL_MAIN(Container *container){
	for(int i=0; i< container->num_d1; i++){
		container->d1[i].print();
	}

	for(int i=0; i< container->num_d2; i++){
		container->d2[i].print();
	}

}

int main(int argc, const char * argv[]){
	hipError_t result = hipSuccess;
	const char *resultString;

	Container *c;

	hipMalloc((void**)&c, sizeof(Container));
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	KERNEL_SETUP<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}
	cout << "SETUP OK..." << std::endl;

	KERNEL_MAIN<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}
	cout << "MAIN OK..." << std::endl;

	KERNEL_CLEANUP<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	if(c != NULL) hipFree(c);
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}
	cout << "CLEANUP OK..." << std::endl;

	// get outta here
	hipDeviceReset();
	return EXIT_SUCCESS;

}
