#include "hip/hip_runtime.h"
// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

// Defs
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

__global__ void KERNEL_SETUP(Container *container){
	container->d1 = new Derived1();
	container->d2 = new Derived2();

}

__global__ void KERNEL_CLEANUP(Container *container){
	if(container->d1 != NULL) delete(d1);
	if(container->d2 != NULL) delete(d2);

}

__global__ void KERNEL_MAIN(Container *container){

}

int main(int argc, const char * argv[]){
	hipError_t result = hipSuccess;
	const char *resultString;

	Container *c;

	hipMalloc((void**)&c, sizeof(Container));
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	// init container with 5 derived classes each
	c->init(5,5);

	KERNEL_SETUP<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	KERNEL_MAIN<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	KERNEL_CLEANUP<<<1,1>>>(c);
	hipDeviceSynchronize();
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	if(c != NULL) hipFree(c);
	result = hipGetLastError();
	if(result != hipSuccess){
		resultString = hipGetErrorString(result);
		cout << "Error : " << resultString << std::endl;
	}

	// get outta here
	hipDeviceReset();
	return EXIT_SUCCESS;

}
