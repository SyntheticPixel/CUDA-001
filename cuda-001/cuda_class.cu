/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_class.h"

using namespace std;

__device__ Base::Base(){

}

__device__ Base::~Base(){
}

__device__ Derived1::Derived1(){
	printf(" Derived1 created\n");

}

__device__ Derived1::~Derived1(){
	printf(" Derived1 deleted\n");

}

__device__ void Derived1::print(int i){
	printf(description);
	printf(" DERIVED_1 : %d\n", i);

}

__device__ Derived2::Derived2(){
	printf(" Derived2 created\n");

}

__device__ Derived2::~Derived2(){
	printf(" Derived2 deleted\n");

}

__device__ void Derived2::print(int i){
	printf(description);
	printf(" DERIVED_2 : %d\n", i);

}

Container::Container(){

	num_d1 = 0;
	num_d2 = 0;

	classes = NULL;
}

Container::~Container(){

}

__device__ void Container::init(int amount1, int amount2){

	// add the claases to the vector
	num_d1 = amount1;
	num_d2 = amount2;

	int t = amount1 + amount2;
	classes = new Base*[t];

	for(int i = 0; i < amount1; i++){
		//classes[i] = reinterpret_cast<Base *>(new Derived1());
		classes[i] = new Derived1();
	}

	for(int i = amount1; i < t; i++){
		//classes[i] = reinterpret_cast<Base *>(new Derived2());
		classes[i] = new Derived2();
	}
}

__device__ void Container::cleanup(){

	// delete all instances
	int t = num_d1 + num_d2;
	for(int i = 0; i < t; i++){
		Base *p;
		//p = reinterpret_cast<Base *>(classes[i]);
		p = classes[i];
		if(p != NULL){
			delete p;
			printf(" deleted pointer %d of %d\n", i, t);
		}
	}

	// delete the master pointer
	if(classes != NULL){
		delete[] classes;
		printf(" deleted master pointer\n");
	}

	num_d1 = 0;
	num_d2 = 0;
}
