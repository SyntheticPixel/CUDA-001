/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

Base::Base(){

}

Base::~Base(){

}

Derived1::Derived1(){

}

Derived1::~Derived1(){

}

void Derived1::print(){
	cout << " DERIVED_1" << std::endl;

}

Derived2::Derived2(){
	cout << " DERIVED_2" << std::endl;

}

Derived2::~Derived2(){

}

void Derived2::print(){

}

Container::Container(){

	num_d1 = 0;
	num_d2 = 0;

	d1 = NULL;
	d2 = NULL;

}

Container::~Container(){

}

Container::init(int d1, int d2){
	num_d1 = d1;
	num_d2 = d2;
}
