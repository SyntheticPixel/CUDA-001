/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

__device__ Base::Base(){
	printf("0");

}

__device__ Base::~Base(){
	printf("0");
}


__device__ void Base::print(int i){
	printf(" BASE : %d\n", i);

}

__device__ Derived1::Derived1(){
	printf("1");

}

__device__ Derived1::~Derived1(){
	printf("-1");

}

__device__ void Derived1::print(int i){
	printf(" DERIVED_1 : %d\n", i);

}

__device__ Derived2::Derived2(){
	printf("2");

}

__device__ Derived2::~Derived2(){
	printf("-2");
}

__device__ void Derived2::print(int i){
	printf(" DERIVED_2 : %d\n", i);

}

Container::Container(){

	num_d1 = 0;
	num_d2 = 0;

	classes = NULL;
}

Container::~Container(){

}

__device__ void Container::init(int amount1, int amount2){

	// add the claases to the vector

	int t = amount1 + amount2;
	classes = new Base*[t];

	for(int i = 0; i < amount1; i++){
		classes[i] = new Derived1();
		printf("\n");
	}

	for(int i = amount1; i < amount1+amount2; i++){
		classes[i] = new Derived2();
		printf("\n");
	}
}

__device__ void Container::cleanup(){

	// delete all instances
	int t = num_d1 + num_d2;
	for(int i = 0; i < t; i++){
		Base * p;
		p = classes[i];
		if(p != NULL) delete p;
	}

	// delete the master pointer
	if(classes != NULL) delete[] classes;

	num_d1 = 0;
	num_d2 = 0;
}
