/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

__device__ Base::Base(){

}

__device__ Base::~Base(){

}

__device__ Derived1::Derived1(){

}

__device__ Derived1::~Derived1(){

}

__device__ void Derived1::print(){
	printf(" DERIVED_1\n");

}

__device__ Derived2::Derived2(){

}

__device__ Derived2::~Derived2(){

}

__device__ void Derived2::print(){
	printf(" DERIVED_2\n");

}

Container::Container(){

	num_d1 = 0;
	num_d2 = 0;

	d1 = NULL;
	d2 = NULL;

}

Container::~Container(){

}

__device__ void Container::init(int amount1, int amount2){
	d1 = new Derived1[amount1];
	d2 = new Derived2[amount2];
}

__device__ void Container::cleanup(){
	if(d1 != NULL) delete[] d1;
	if(d2 != NULL) delete[] d2;
}
