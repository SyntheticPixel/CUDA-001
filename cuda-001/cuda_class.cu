/*
 *
 */

// Includes
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "cuda_class.h"

using namespace std;

__device__ Base::Base(){

}

__device__ Base::~Base(){

}

/*
__device__ void Base::print(int i){
	printf(" BASE : %d\n", i);

}
*/

__device__ Derived1::Derived1(){
	printf("1");

}

__device__ Derived1::~Derived1(){

}

__device__ void Derived1::print(int i){
	printf(" DERIVED_1 : %d\n", i);

}

__device__ Derived2::Derived2(){
	printf("2");

}

__device__ Derived2::~Derived2(){
}

__device__ void Derived2::print(int i){
	printf(" DERIVED_2 : %d\n", i);

}

Container::Container(){

	num_d1 = 0;
	num_d2 = 0;

	d1 = NULL;
	d2 = NULL;

}

Container::~Container(){

}

__device__ void Container::init(int amount1, int amount2){
	num_d1 = amount1;
	d1 = new Derived1[amount1];
	printf("\n");

	num_d2 = amount2;
	d2 = new Derived2[amount2];
	printf("\n");
}

__device__ void Container::cleanup(){
	if(d1 != NULL) delete[] d1;
	if(d2 != NULL) delete[] d2;

	num_d1 = 0;
	num_d2 = 0;
}
